#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_WARNINGS

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <stdint.h>

#include "prime-gmp.h"

typedef uint32_t uint;
typedef uint64_t ulong;

#define MAX_N_SIZE 128
#define MAX_BLOCK_SIZE 32
#define MAX_JOB_SIZE 2048

template<uint N_Size>
__global__
void fermat_test(const uint *M_in, const uint *Mi_in, const uint *R_in, uint *is_prime) {

	uint R[N_Size];
	uint M[N_Size];

	{
		// Get the index of the current element to be processed
		const int offset = (blockDim.x*blockIdx.x + threadIdx.x) * N_Size;

		for (int i = 0; i < N_Size; ++i)
		{
			M[i] = M_in[offset + i];
			R[i] = R_in[offset + i];
		}
	}

	const uint shift = __clz(M[N_Size - 1]);
	const uint highbit = ((uint)1) << 31;
	uint startbit = highbit >> shift;

	const uint mi = Mi_in[blockDim.x*blockIdx.x + threadIdx.x];

	int en = N_Size;
#pragma unroll 1
	while (en-- > 0)
	{
		uint bit = startbit;
		startbit = highbit;
		uint E = M[en];
		if (en == 0) E--;

		do
		{
			{
				uint P[N_Size * 2];
				//mpn_sqr(pp, rp, mn);
				P[0] = R[0] * R[0];
				P[1] = __umulhi(R[0], R[0]);
				{
					uint T[N_Size * 2];

					{
						uint cy = 0;
						for (int i = 0; i < N_Size - 1; ++i)
						{
							T[i] = R[i + 1] * R[0];
							T[i] += cy;
							cy = (T[i] < cy) + __umulhi(R[i + 1], R[0]);
						}
						T[N_Size - 1] = cy;
					}

#pragma unroll 1
					for (int j = 2; j < N_Size; ++j)
					{
						uint cy = 0;
						for (int i = j; i < N_Size; ++i)
						{
							uint lp = R[i] * R[j - 1];
							lp += cy;
							cy = (lp < cy) + __umulhi(R[i], R[j - 1]);
							lp += T[i + j - 2];
							cy += lp < T[i + j - 2];
							T[i + j - 2] = lp;
						}
						T[N_Size + j - 2] = cy;
					}

					for (int i = 0; i < N_Size; ++i)
					{
						P[2 * i] = R[i] * R[i];
						P[2 * i + 1] = __umulhi(R[i], R[i]);
					}

					uint cy = 0;
					for (int i = 0; i < N_Size * 2 - 2; ++i)
					{
						uint t = T[i] & highbit;
						T[i] <<= 1;
						T[i] |= cy;
						cy = t >> 31;
					}
					P[2 * N_Size - 1] += cy;

					cy = 0;
					for (int i = 0; i < N_Size * 2 - 2; ++i)
					{
						uint a = P[i + 1] + cy;
						cy = (a < P[i + 1]);
						a += T[i];
						cy += (a < T[i]);
						P[i + 1] = a;
					}
					P[2 * N_Size - 1] += cy;
				}

				//if (mpn_redc_1(rp, pp, mp, mn, mi) != 0) 
				//  mpn_sub_n(rp, rp, mshifted, n);
#pragma unroll 1
				for (int j = 0; j < N_Size; ++j)
				{
					uint cy = 0;
					uint v = P[j] * mi;
					for (int i = 0; i < N_Size; ++i)
					{
						uint lp = M[i] * v;
						lp += cy;
						cy = (lp < cy) + __umulhi(M[i], v);
						lp += P[i + j];
						cy += lp < P[i + j];
						P[i + j] = lp;
					}
					R[j] = cy;
				}

				{
					uint cy = 0;
					for (int i = 0; i < N_Size; ++i)
					{
						uint a = R[i] + cy;
						cy = (a < R[i]);
						a += P[i + N_Size];
						cy += (a < P[i + N_Size]);
						R[i] = a;
					}

					if (cy != 0)
					{
						cy = 0;
						uint last_shifted = 0;
						for (int i = 0; i < N_Size; ++i)
						{
							uint a = R[i];
							uint b = (M[i] << shift) | last_shifted;
							last_shifted = M[i] >> (32 - shift);
							b += cy;
							cy = (b < cy);
							cy += (a < b);
							R[i] = a - b;
						}
					}
				}
			}

			if (E & bit)
			{
				//mp_limb_t carry = mpn_lshift(rp, rp, mn, 1);
				uint carry = 0;
				for (int i = 0; i < N_Size; ++i)
				{
					uint t = R[i] & highbit;
					R[i] <<= 1;
					R[i] |= carry;
					carry = t >> 31;
				}
				while (carry)
				{
					//carry -= mpn_sub_n(rp, rp, mshifted, mn);
					uint cy = 0;
					uint last_shifted = 0;
					for (int i = 0; i < N_Size; ++i)
					{
						uint a = R[i];
						uint b = (M[i] << shift) | last_shifted;
						last_shifted = M[i] >> (32 - shift);
						b += cy;
						cy = (b < cy);
						cy += (a < b);
						R[i] = a - b;
					}
					carry -= cy;
				}
			}
			bit >>= 1;
		} while (bit > 0);

	}

	// DeREDCify - necessary as rp can have a large
	//             multiple of m in it (although I'm not 100% sure
	//             why it can't after this redc!)
	{
		uint T[N_Size * 2];
		for (int i = 0; i < N_Size; ++i)
		{
			T[i] = R[i];
			T[N_Size + i] = 0;
		}

		// MPN_REDC_1(rp, tp, mp, mn, mi);
#pragma unroll 1
		for (int j = 0; j < N_Size; ++j)
		{
			uint cy = 0;
			uint v = T[j] * mi;
			for (int i = 0; i < N_Size; ++i)
			{
				uint lp = M[i] * v;
				lp += cy;
				cy = (lp < cy) + __umulhi(M[i], v);
				lp += T[i + j];
				cy += lp < T[i + j];
				T[i + j] = lp;
			}
			R[j] = cy;
		}

		{
			uint cy = 0;
			for (int i = 0; i < N_Size; ++i)
			{
				uint a = R[i] + cy;
				cy = (a < R[i]);
				a += T[i + N_Size];
				cy += (a < T[i + N_Size]);
				R[i] = a;
			}

			if (cy != 0)
			{
				cy = 0;
				uint last_shifted = 0;
				for (int i = 0; i < N_Size; ++i)
				{
					uint a = R[i];
					uint b = (M[i] << shift) | last_shifted;
					last_shifted = M[i] >> (32 - shift);
					b += cy;
					cy = (b < cy);
					cy += (a < b);
					R[i] = a - b;
				}
			}
		}
	}

	bool result = true;
	if (R[N_Size - 1] != 0)
	{
		// Compare to m+1
		uint cy = 1;
		for (int i = 0; i < N_Size && result; ++i)
		{
			uint a = M[i] + cy;
			cy = a < M[i];
			if (R[i] != a) result = false;
		}
	}
	else
	{
		// Compare to 1
		result = R[0] == 1;
		for (int i = 1; i < N_Size && result; ++i)
		{
			if (R[i] != 0) result = false;
		}
	}

	is_prime[blockDim.x*blockIdx.x + threadIdx.x] = result;
}

#define DEBUG 0

#define MAX_SOURCE_SIZE (0x100000)

const unsigned char  binvert_limb_table[128] = {
	0x01, 0xAB, 0xCD, 0xB7, 0x39, 0xA3, 0xC5, 0xEF,
	0xF1, 0x1B, 0x3D, 0xA7, 0x29, 0x13, 0x35, 0xDF,
	0xE1, 0x8B, 0xAD, 0x97, 0x19, 0x83, 0xA5, 0xCF,
	0xD1, 0xFB, 0x1D, 0x87, 0x09, 0xF3, 0x15, 0xBF,
	0xC1, 0x6B, 0x8D, 0x77, 0xF9, 0x63, 0x85, 0xAF,
	0xB1, 0xDB, 0xFD, 0x67, 0xE9, 0xD3, 0xF5, 0x9F,
	0xA1, 0x4B, 0x6D, 0x57, 0xD9, 0x43, 0x65, 0x8F,
	0x91, 0xBB, 0xDD, 0x47, 0xC9, 0xB3, 0xD5, 0x7F,
	0x81, 0x2B, 0x4D, 0x37, 0xB9, 0x23, 0x45, 0x6F,
	0x71, 0x9B, 0xBD, 0x27, 0xA9, 0x93, 0xB5, 0x5F,
	0x61, 0x0B, 0x2D, 0x17, 0x99, 0x03, 0x25, 0x4F,
	0x51, 0x7B, 0x9D, 0x07, 0x89, 0x73, 0x95, 0x3F,
	0x41, 0xEB, 0x0D, 0xF7, 0x79, 0xE3, 0x05, 0x2F,
	0x31, 0x5B, 0x7D, 0xE7, 0x69, 0x53, 0x75, 0x1F,
	0x21, 0xCB, 0xED, 0xD7, 0x59, 0xC3, 0xE5, 0x0F,
	0x11, 0x3B, 0x5D, 0xC7, 0x49, 0x33, 0x55, 0xFF
};

#define binvert_limb(inv,n)                                             \
  do {                                                                  \
    mp_limb_t  __n = (n);                                               \
    mp_limb_t  __inv;                                                   \
    assert ((__n & 1) == 1);                                            \
                                                                        \
    __inv = binvert_limb_table[(__n/2) & 0x7F]; /*  8 */                \
    if (GMP_LIMB_BITS > 8)   __inv = 2 * __inv - __inv * __inv * __n;   \
    if (GMP_LIMB_BITS > 16)  __inv = 2 * __inv - __inv * __inv * __n;   \
    if (GMP_LIMB_BITS > 32)  __inv = 2 * __inv - __inv * __inv * __n;   \
                                                                        \
    assert ((__inv * __n) == 1);                        \
    (inv) = __inv;                                      \
  } while (0)

static void setup_fermat(int N_Size, int num, const mp_limb_t* M, mp_limb_t* MI, mp_limb_t* R)
{
	assert(N_Size <= MAX_N_SIZE);
	for (int j = 0; j < num; ++j)
	{
		mp_size_t mn = N_Size;
		mp_limb_t mshifted[MAX_N_SIZE];
		mp_srcptr mp;
		mp_ptr rp;
		struct gmp_div_inverse minv;

		// REDCify: r = B^n * 2 % M
		mp = &M[j*N_Size];
		rp = &R[j*N_Size];
		mpn_div_qr_invert(&minv, mp, mn);

		if (minv.shift > 0)
		{
			mpn_lshift(mshifted, mp, mn, minv.shift);
			mp = mshifted;
		}
		else
		{
			for (size_t i = 0; i < mn; ++i) mshifted[i] = mp[i];
		}

		for (size_t i = 0; i < mn; ++i) rp[i] = 0;
		rp[mn] = 1 << minv.shift;
		mpn_div_r_preinv_ns(rp, mn + 1, mp, mn, &minv);

		if (minv.shift > 0)
		{
			mpn_rshift(rp, rp, mn, minv.shift);
			mp = &M[j*N_Size];
		}

		mp_limb_t mi;
		binvert_limb(mi, mp[0]);
		MI[j] = -mi;
	}
}

#if DEBUG
#define DPRINTF(fmt, args...) do { printf("line %d: " fmt, __LINE__, ##args); fflush(stdout); } while(0)
#else
#define DPRINTF(fmt, ...) do { } while(0)
#endif

typedef struct PrimeTestCxt
{
	uint* m_mem_obj;
	uint* mi_mem_obj;
	uint* r_mem_obj;
	uint* is_prime_mem_obj;

	uint *R;
	uint *MI;
} PrimeTestCxt;

PrimeTestCxt* primeTestInit()
{
	hipError_t cudaStatus;

	PrimeTestCxt* cxt = (PrimeTestCxt*)malloc(sizeof(PrimeTestCxt));

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return NULL;
	}

	// Create memory buffers on the device for each vector 
	cudaStatus = hipMalloc((void**)&cxt->m_mem_obj, MAX_JOB_SIZE * MAX_N_SIZE * sizeof(uint));
	cudaStatus = hipMalloc((void**)&cxt->mi_mem_obj, MAX_JOB_SIZE * sizeof(uint));
	cudaStatus = hipMalloc((void**)&cxt->r_mem_obj, MAX_JOB_SIZE * MAX_N_SIZE * sizeof(uint));
	cudaStatus = hipMalloc((void**)&cxt->is_prime_mem_obj, MAX_JOB_SIZE * sizeof(uint));

	// Create buffers on host
	cxt->R = (uint*)malloc(sizeof(uint)*(MAX_N_SIZE*MAX_JOB_SIZE + 1));
	cxt->MI = (uint*)malloc(sizeof(uint)*MAX_JOB_SIZE);

	return cxt;
}

void primeTest(PrimeTestCxt* cxt, int N_Size, int listSize, const uint* M, uint* is_prime)
{
	hipError_t cudaStatus;

	int nextJobSize = min(MAX_JOB_SIZE, listSize);

	if (nextJobSize > 0)
	{
		setup_fermat(N_Size, nextJobSize, M, cxt->MI, cxt->R);
	}

	while (nextJobSize > 0)
	{
		int jobSize = nextJobSize;
		listSize -= jobSize;
		nextJobSize = min(MAX_JOB_SIZE, listSize);

		// Copy the lists A and B to their respective memory buffers
		cudaStatus = hipMemcpy(cxt->mi_mem_obj, cxt->MI, jobSize * sizeof(uint), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(cxt->r_mem_obj, cxt->R, jobSize * N_Size * sizeof(uint), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(cxt->m_mem_obj, M, jobSize * N_Size * sizeof(uint), hipMemcpyHostToDevice);

		int blockSize = 1;
		int numBlocks = jobSize;
		while (blockSize < MAX_BLOCK_SIZE && ((numBlocks & 1) == 0))
		{
			numBlocks >>= 1;
			blockSize <<= 1;
		}

		DPRINTF("before execution\n");
		switch (N_Size)
		{
		case 3: fermat_test<3> << <numBlocks, blockSize >> >(cxt->m_mem_obj, cxt->mi_mem_obj, cxt->r_mem_obj, cxt->is_prime_mem_obj); break;
		case 20: fermat_test<20> << <numBlocks, blockSize >> >(cxt->m_mem_obj, cxt->mi_mem_obj, cxt->r_mem_obj, cxt->is_prime_mem_obj); break;
		case 40: fermat_test<40> << <numBlocks, blockSize >> >(cxt->m_mem_obj, cxt->mi_mem_obj, cxt->r_mem_obj, cxt->is_prime_mem_obj); break;
		case 53: fermat_test<53> <<<numBlocks, blockSize >>>(cxt->m_mem_obj, cxt->mi_mem_obj, cxt->r_mem_obj, cxt->is_prime_mem_obj); break;
		default: abort();
		}
		
		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			printf("addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return;
		}

		if (nextJobSize > 0)
		{
			M += jobSize*N_Size;
			setup_fermat(N_Size, nextJobSize, M, cxt->MI, cxt->R);
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			printf("hipDeviceSynchronize returned error code %d after launching kernel!\n", cudaStatus);
			return;
		}

		cudaStatus = hipMemcpy(is_prime, cxt->is_prime_mem_obj, jobSize * sizeof(uint), hipMemcpyDeviceToHost);

		is_prime += jobSize;
	}
}

void primeTestTerm(PrimeTestCxt* cxt)
{
	hipFree(cxt->mi_mem_obj);
	hipFree(cxt->m_mem_obj);
	hipFree(cxt->r_mem_obj);
	hipFree(cxt->is_prime_mem_obj);

	free(cxt->R);
	free(cxt->MI);
	free(cxt);

	hipDeviceReset();
}